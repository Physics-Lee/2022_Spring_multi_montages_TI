#include "hip/hip_runtime.h"
#pragma once
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
// MATLAB related
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "mxShowCriticalErrorMessage.c"
#include "MultiElecNtKernel.cuh"

#define	NE_MX	prhs[0]
#define	E0	    prhs[1]
#define	ELECA 	prhs[2]
#define	ELECB 	prhs[3]
#define	CUA	    prhs[4]
#define	CUB	    prhs[5]
#define	VOLUME	prhs[6]
#define	THRES	prhs[7]
#define	ALPHA	prhs[8]

#define	K_MX	prhs[9]
#define	VFLAG	prhs[10]
#define	BLOCKSIZE	prhs[11]

#define	RETVAL1	plhs[0]
#define	RETVAL2	plhs[1]
// #define	RETVAL3	plhs[2]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
    // -------------------------------------------------------------------------
    // vflag, blockSize
    // -------------------------------------------------------------------------
    bool vflag = *(bool*)mxGetData(VFLAG);
    int blockSize = *(int*)mxGetData(BLOCKSIZE);
    // -------------------------------------------------------------------------
    // initial
    // -------------------------------------------------------------------------
    mxInitGPU();
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    float time, timePrepare, timeKernel, timeRest;
    timeKernel = 0;
    timeRest = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    const mwSize ndim = 1;
    mwSize dims[ndim];  
    // -------------------------------------------------------------------------
    // N, N128, E0
    // -------------------------------------------------------------------------
    hipEventRecord(start, 0);
    int N = *(int*)mxGetData(NE_MX); 
    mxGPUArray const *e0 = mxGPUCreateFromMxArray(E0);
    float *d_e0 = (float*)mxGPUGetDataReadOnly(e0);
    const mwSize *dim0 = mxGetDimensions(E0);
    const int L = dim0[1];
    const int N128 = dim0[0]; 
    // -------------------------------------------------------------------------
    // electrode
    // -------------------------------------------------------------------------
    mxGPUArray const *elecA = mxGPUCreateFromMxArray(ELECA);
    mxGPUArray const *elecB = mxGPUCreateFromMxArray(ELECB);
    int Nm = mxGetM(ELECA);
    int elecNumA = mxGetN(ELECA);
    int elecNumB = mxGetN(ELECB);
    int * d_elecA = (int*)mxGPUGetDataReadOnly(elecA);
    int * d_elecB = (int*)mxGPUGetDataReadOnly(elecB);
    // -------------------------------------------------------------------------
    // current
    // -------------------------------------------------------------------------
    mxGPUArray const *cuA = mxGPUCreateFromMxArray(CUA);
    float * d_cuA = (float*)mxGPUGetDataReadOnly(cuA);
    mxGPUArray const *cuB = mxGPUCreateFromMxArray(CUB);
    float * d_cuB = (float*)mxGPUGetDataReadOnly(cuB);
    // -------------------------------------------------------------------------
    // volume
    // -------------------------------------------------------------------------
    mxGPUArray const *volume = mxGPUCreateFromMxArray(VOLUME);
    float *d_volume = (float*)mxGPUGetDataReadOnly(volume);
    int Nvolume = mxGetM(VOLUME);
    // -------------------------------------------------------------------------
    // thres, alpha
    // -------------------------------------------------------------------------
    float thres = *(float*)mxGetData(THRES); 
    int alpha = *(int*)mxGetData(ALPHA); 
    // -------------------------------------------------------------------------
    // K loop num
    // -------------------------------------------------------------------------
    int K = *(int*)mxGetData(K_MX);
    // -------------------------------------------------------------------------
    // display fundimental data information
    // -------------------------------------------------------------------------
    if(vflag){
        mexPrintf("Leadfield element number: %d\n", N); ;
        mexPrintf("Leadfield element number with padding: %d\n", N128);
        mexPrintf("Electrode pool number: %d\n", L);
        mexPrintf("Montage number: %d\n", Nm);
        mexPrintf("Elec A Number: %d\n", elecNumA);
        mexPrintf("Elec B Number: %d\n", elecNumB);
        mexPrintf("Volume element Number: %d\n", Nvolume);
        switch(alpha){
            case -1:
                mexPrintf("Output is the MAX value in the region.\n");
                break;
            case 0:
                mexPrintf("Output is VOLUME above thres in the region.\n");
                break;
            default:
                mexPrintf("Output is Volume Weighted Summation in the region.\n");
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        timePrepare = time;
        mexPrintf("prepare time:  %3.3f ms \n",timePrepare);
        mexEvalString("drawnow") ;
    }
    // =========================================================================
    // predefine return r1 and internal r2
    // =========================================================================
    dims[0] = Nm;
    mxGPUArray * r1 = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (r1==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");
    float *d_r1 = (float*)mxGPUGetData(r1);  
    int r2Size = N128/blockSize*K;
    dims[0] = r2Size;
    mxGPUArray * r2 = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (r2==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");
    float *d_r2 = (float*)mxGPUGetData(r2);
    if(vflag)mexPrintf("r2Size:  %d\n", r2Size);
    // -------------------------------------------------------------------------
    // interval r3 is for check
    // -------------------------------------------------------------------------
    // int r3Size = N128*K;
    // dims[0] = r3Size;
    // mxGPUArray * r3 = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    // if (r3==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");
    // float *d_r3 = (float*)mxGPUGetData(r3);
    // if(vflag)mexPrintf("r3Size:  %d\n", r3Size);
    // =========================================================================
    // process
    // =========================================================================
    int Nloop = Nm/K+1;
    int Nlast = Nm % K;
    int N1 = N128 / blockSize;
    int loop100 = ceil((float)Nloop/100);
    if(vflag){
        mexPrintf("Nloop:  %d, K: %d \n", Nloop,K);
        mexEvalString("drawnow") ;
    }
    int Ki = 0;
    for (int i = 0; i<Nloop; i++){
        if(vflag)hipEventRecord(start, 0);
        if(i<Nloop-1) Ki = K;
        else Ki = Nlast;
        if (Ki == 0) break;
        int Mbase = i*K;  
        int gridSize = N128/blockSize*Ki;
        if(i==0){
            if(vflag){
                mexPrintf("blockSize:  %d, gridSize: %d\n", blockSize, gridSize);
                mexEvalString("drawnow") ;
            }
        }
        if(i==loop100){
            if(vflag){
                mexPrintf("The time consumption of %d loops is about %3.3f ms \n", loop100, timeKernel + timeRest);
                mexEvalString("drawnow") ;
            } 
        }
        unsigned sharedMemSize = blockSize*sizeof(float);
        switch(alpha){
            case -1:
                MultiMaxNtKernel<<<gridSize, blockSize, sharedMemSize>>>(N128, d_e0, Nm, elecNumA, elecNumB, d_elecA, d_elecB, d_cuA, d_cuB, Mbase, d_r2);
                break;
            case 0:
                MultiVolumeNtKernel<<<gridSize, blockSize, sharedMemSize>>>(N128, d_e0, Nm, elecNumA, elecNumB, d_elecA, d_elecB, d_cuA, d_cuB, Mbase, d_volume, thres, d_r2);
                break;
            default:
                MultiAlphaNtKernel<<<gridSize, blockSize, sharedMemSize>>>(N128, d_e0, Nm, elecNumA, elecNumB, d_elecA, d_elecB, d_cuA, d_cuB, Mbase, d_volume, alpha, d_r2);
        }
        hipDeviceSynchronize();   
        if(vflag){ 
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
            timeKernel += time;
            // mexPrintf("Kernel function finished.\n");
            // mexEvalString("drawnow") ;
        }  
        // if(i==0)RETVAL2 = mxGPUCreateMxArrayOnCPU(r2);
        // if(i==0)RETVAL3 = mxGPUCreateMxArrayOnCPU(r3);
        // ---------------------------------------------------------------------
        // Rest reduction
        // ---------------------------------------------------------------------
        if(vflag)hipEventRecord(start, 0);        
        int blockSize1 = 8;
        int gridSize1 = (Ki + blockSize1 - 1) / blockSize1;
        if(alpha>-1)getSum<<<gridSize1,blockSize1>>>(Ki, N1, d_r2, d_r1+i*K);
        else getMax<<<gridSize1,blockSize1>>>(Ki, N1, d_r2, d_r1+i*K);
        hipDeviceSynchronize();
        if(vflag){ 
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
            timeRest += time;
        }  
    }   
    // divide sum volume
    if(alpha>-1){
    float volumeSum;
    hipblasStatus_t = hipblasSasum(cublasHandle, N, d_volume, 1, &volumeSum);
    if (hipblasStatus_t!= HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage("hipblasSasum in volume failed");
    volumeSum = 1/volumeSum;
    hipblasStatus_t = hipblasSscal(cublasHandle, Nm, &volumeSum, d_r1, 1);
    if (hipblasStatus_t!= HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage("hipblasSscal in volume scale failed");
    }
    RETVAL1 = mxGPUCreateMxArrayOnCPU(r1);
    // =========================================================================
    // destroy
    // =========================================================================
    if(vflag){
        mexPrintf("Kernel time:  %3.3f ms \n",timeKernel);
        mexEvalString("drawnow") ;
        mexPrintf("Rest PR time:  %3.3f ms \n",timeRest);
        mexEvalString("drawnow") ;
        mexPrintf("GPU part with NT end...\n");
    }
    // mxGPUDestroyGPUArray(r3);
    mxGPUDestroyGPUArray(r2);
    mxGPUDestroyGPUArray(r1);
    mxGPUDestroyGPUArray(volume);
    mxGPUDestroyGPUArray(elecA);
    mxGPUDestroyGPUArray(elecB);
    mxGPUDestroyGPUArray(cuA);
    mxGPUDestroyGPUArray(cuB);
    mxGPUDestroyGPUArray(e0);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(cublasHandle);
}
  