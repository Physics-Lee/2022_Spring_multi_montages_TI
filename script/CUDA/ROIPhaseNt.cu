#include <stdlib.h>
#include <stdio.h>
#include <string.h>
/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
// MATLAB related
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "mxShowCriticalErrorMessage.c"
#include "ROIKernel.cuh"

#define	NE_MX	prhs[0]
#define	E0	    prhs[1]
#define	ELEC	prhs[2]
#define	CU	    prhs[3]
#define	VOLUME	prhs[4]
#define	THRES	prhs[5]
#define	ALPHA	prhs[6]

#define	K_MX	prhs[7]
#define	VFLAG	prhs[8]
#define	BLOCKSIZE	prhs[9]

#define	RETVAL1	plhs[0]
// #define	RETVAL2	plhs[1]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
    // =========================================================================
    // initial
    // =========================================================================
    mxInitGPU();
    float time, timePrepare, timeKernel, timeRest;
    timeKernel = 0;
    timeRest = 0;
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    hipError_t cudaStatus;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    const mwSize ndim = 1;
    mwSize dims[ndim];
    // -------------------------------------------------------------------------
    // vflag, blockSize
    // -------------------------------------------------------------------------
    bool vflag = *(bool*)mxGetData(VFLAG);
    int blockSize = *(int*)mxGetData(BLOCKSIZE);
    // =========================================================================
    // N, N128, E0
    // =========================================================================
    if (vflag)hipEventRecord(start, 0);
    int N = *(int*)mxGetData(NE_MX); 
    mxGPUArray const *e0 = mxGPUCreateFromMxArray(E0);
    float *d_e0 = (float*)mxGPUGetDataReadOnly(e0);
    const mwSize *dim0 = mxGetDimensions(E0);
    const int L = dim0[1];
    const int N128 = dim0[0]; 
    // -------------------------------------------------------------------------
    // electrode
    // -------------------------------------------------------------------------
    mxGPUArray const *elec = mxGPUCreateFromMxArray(ELEC);
    int Nelec = mxGetM(ELEC);
    int * d_elec = (int*)mxGPUGetDataReadOnly(elec);
    // -------------------------------------------------------------------------
    // current
    // -------------------------------------------------------------------------
    mxGPUArray const *cu = mxGPUCreateFromMxArray(CU);
    float* d_cu = (float*)mxGPUGetDataReadOnly(cu);  
    int Ncu = (int)mxGetNumberOfElements(CU);
    int Nm = Ncu * Nelec;
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cuConst), d_cu, sizeof(float) * Ncu);
    if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("hipMemcpyToSymbol failed");
    // -------------------------------------------------------------------------
    // volume
    // -------------------------------------------------------------------------// =========================================================================
    mxGPUArray const *volume = mxGPUCreateFromMxArray(VOLUME);
    float *d_volume = (float*)mxGPUGetDataReadOnly(volume);
    int Nvolume = mxGetM(VOLUME);
    // -------------------------------------------------------------------------
    // thres, alpha
    // -------------------------------------------------------------------------
    float thres = *(float*)mxGetData(THRES); 
    int alpha = *(int*)mxGetData(ALPHA); 
    // -------------------------------------------------------------------------
    // K loop num
    // -------------------------------------------------------------------------
    int K = *(int*)mxGetData(K_MX); 
    // -------------------------------------------------------------------------
    // display fundimental data information
    // -------------------------------------------------------------------------
    if(vflag){
        mexPrintf("Leadfield element number: %d\n", N); ;
        mexPrintf("Leadfield element number with padding: %d\n", N128);
        mexPrintf("Volume element Number: %d\n", Nvolume);
        mexPrintf("Electrode pool number: %d\n", L);
        mexPrintf("Electrode combination number: %d\n", Nelec);
        mexPrintf("Current type number: %d\n", Ncu);
        mexPrintf("Montage number: %d\n", Nm);
        mexPrintf("threshold: %3.3f\n", thres);
        switch(alpha){
            case -1:
                mexPrintf("Output is the MAX value in the region.\n");
                break;
            case 0:
                mexPrintf("Output is VOLUME above thres in the region.\n");
                break;
            default:
                mexPrintf("Output is Volume Weighted Summation in the region.\n");
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        timePrepare = time;
        mexPrintf("prepare time:  %3.3f ms \n",timePrepare);
        mexEvalString("drawnow") ;
    }
    // =========================================================================
    // predefine return r1 and internal r2
    // =========================================================================
    dims[0] = Nm;
    mxGPUArray * r1 = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (r1==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");
    float *d_r1 = (float*)mxGPUGetData(r1);
    int r2Size = N128/blockSize*K;
    dims[0] = r2Size;
    mxGPUArray * r2 = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (r2==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");
    float *d_r2 = (float*)mxGPUGetData(r2);
    // -------------------------------------------------------------------------
    // interval r3 is for check
    // -------------------------------------------------------------------------
    int r3Size = N128*K;
    dims[0] = r3Size;
    mxGPUArray * r3 = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (r3==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");
    float *d_r3 = (float*)mxGPUGetData(r3);
    if(vflag)mexPrintf("r3Size:  %d\n", r3Size);
    // =========================================================================
    // process
    // =========================================================================
    int Nloop = Nelec/K+1;
    int Nlast = Nelec % K;
    int Ki = 0;
    for (int i = 0; i<Nloop; i++){
        if(vflag)hipEventRecord(start, 0);
        if(i<Nloop-1) Ki = K;
        else Ki = Nlast;
        if (Ki == 0) break;
        int Cbase = i*K;
        int N1 = N128 / blockSize;
        int gridSize = N128/blockSize*Ki;
        if(i==0){
            if(vflag){
                mexPrintf("blockSize:  %d, gridSize: %d \n", blockSize,gridSize);
                mexPrintf("Nloop: %d, Ki:  %d\n", Nloop, Ki);
                mexEvalString("drawnow") ;
            }
        }
        unsigned sharedMemSize = blockSize * sizeof(float);
        switch(alpha){
            case -1:
                ROIMaxNtKernel<<<gridSize, blockSize,sharedMemSize>>>(Ki, Cbase, N128, d_e0, Nelec, d_elec, Ncu, d_r2);
                break;
            case 0:
                ROIVolumeNtKernel<<<gridSize, blockSize,sharedMemSize>>>(Ki, Cbase, N128, d_e0, Nelec, d_elec, Ncu, d_volume, thres, d_r2);
                break;
            default:
                ROIAlphaNtKernel<<<gridSize, blockSize,sharedMemSize>>>(Ki, Cbase, N128, d_e0, Nelec, d_elec, Ncu, d_volume, alpha, d_r2);
        }
        hipDeviceSynchronize();   
        if(vflag){ 
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
            timeKernel += time;
        }
        // if(i==0)RETVAL2 = mxGPUCreateMxArrayOnCPU(r2); 
        // =====================================================================
        // Rest reduction
        // =====================================================================
        int M = Ki*Ncu;
        if(vflag)hipEventRecord(start, 0); 
        int blockSize1 = 8;
        int gridSize1 = (M + blockSize1 - 1) / blockSize1;
        if(alpha>-1)getSum<<<gridSize1,blockSize1>>>(N1, M, Ki, Cbase, Nelec, d_r2, d_r1);
        else getMax<<<gridSize1,blockSize1>>>(N1, M, Ki, Cbase, Nelec, d_r2, d_r1);  
        hipDeviceSynchronize();
        if (vflag){
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
            timeRest += time;
        }
    }
    // -------------------------------------------------------------------------
    // divide sum volume
    // -------------------------------------------------------------------------
    if(alpha>-1){
        float volumeSum;
        hipblasStatus_t = hipblasSasum(cublasHandle, N, d_volume, 1, &volumeSum);
        if (hipblasStatus_t!= HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage("hipblasSasum in volume failed");
        volumeSum = 1/volumeSum;
        hipblasStatus_t = hipblasSscal(cublasHandle, Nm, &volumeSum, d_r1, 1);
        if (hipblasStatus_t!= HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage("hipblasSscal in volume scale failed");
    }
    // =========================================================================
    // output
    // =========================================================================
    RETVAL1 = mxGPUCreateMxArrayOnCPU(r1);
    // =========================================================================
    // destroy
    // =========================================================================
    mxGPUDestroyGPUArray(r2);
    mxGPUDestroyGPUArray(r1);
    mxGPUDestroyGPUArray(e0);
    mxGPUDestroyGPUArray(elec);
    mxGPUDestroyGPUArray(cu);
    mxGPUDestroyGPUArray(volume);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(cublasHandle);
    if(vflag){
        mexPrintf("Kernel time:  %3.3f ms \n",timeKernel);
        mexEvalString("drawnow") ;
        mexPrintf("Rest reduction time:  %3.3f ms \n",timeRest);
        mexEvalString("drawnow") ;
        mexPrintf("ROI with NT phase GPU part end...\n");
    }
}
  